#include "hip/hip_runtime.h"
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "monte_carlo.h"


__global__ void pi_estimator_kernel(hiprandState_t *states, int *block_counts, const long long NUM_POINTS) {
    extern __shared__ int s_counts[]; // dynamic shared memory (across thread blocks)
    int tid = threadIdx.x;
    // global index of this thread
    long long idx = (long long)blockDim.x * blockIdx.x + tid;


    int local_count = 0;
    if (idx < NUM_POINTS) {
        // produce random point
        float x = hiprand_uniform(&states[idx]);  
        float y = hiprand_uniform(&states[idx]);  
        if (x * x + y * y <= 1.0f) {
            local_count = 1;
        }
    }

    // store each thread's local count into shared memory
    s_counts[tid] = local_count;
    __syncthreads(); // synchronization point

    // perform a tree-reduce in shared memory to sum up block-level counts
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            s_counts[tid] += s_counts[tid + stride];
        }
        __syncthreads();
    }

    // thread 0 of each block writes the block’s total into global memory (vram)
    if (tid == 0) {
        block_counts[blockIdx.x] = s_counts[0];
    }
}
