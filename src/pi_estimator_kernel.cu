#include "hip/hip_runtime.h"
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "monte_carlo.h"

// setup hiprand states
__global__ void setup_curand_states(hiprandState_t *states, unsigned long seed, const long long NUM_POINTS) {
    long long idx = (long long)blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= NUM_POINTS) return;
    hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void pi_estimator_kernel(hiprandState_t *states, int *block_counts, const long long NUM_POINTS) {
    extern __shared__ int s_counts[];

    int tid = threadIdx.x;
    long long idx = (long long)blockDim.x * blockIdx.x + tid;

    int local_count = 0;
    if (idx < NUM_POINTS) {
        float x = hiprand_uniform(&states[idx]);  
        float y = hiprand_uniform(&states[idx]);  
        if (x * x + y * y <= 1.0f) {
            local_count = 1;
        }
    }

    // write into shared memory
    s_counts[tid] = local_count;
    __syncthreads();

    // tree-reduce in shared memory
    // half the threads add pairwise
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            s_counts[tid] += s_counts[tid + stride];
        }
        __syncthreads();
    }

    // thread 0 of each block writes the block’s total into global memory
    if (tid == 0) {
        block_counts[blockIdx.x] = s_counts[0];
    }
}
