#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include "monte_carlo.h"

__global__ void setup_curand_states(hiprandState_t *blockStates, unsigned long seed) {
    int b = blockIdx.x;
    if (threadIdx.x == 0) {
        hiprand_init(seed, b, 0, &blockStates[b]);
    }
}
